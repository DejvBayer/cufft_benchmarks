
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>

#include <hipfft/hipfftXt.h>
#include <hiprand/hiprand.h>
#include <hip/hip_runtime_api.h>

#define checkCudaError(error)                                      \
  if ((error) != hipSuccess)                                      \
  {                                                                \
    std::cout << "CUDA Error: " << hipGetLastError()         \
              << " (File" << __FILE__ << ", line" << __LINE__ << ")" << std::endl; \
  }
#define checkCuFFTError(error)                                                 \
  if ((error) != HIPFFT_SUCCESS)                                                \
  {                                                                            \
    std::cout << "cuFFT Error: " << std::hex << static_cast<int>(error) << std::dec << " (File" << __FILE__ << ", line" << __LINE__ << ")" << std::endl; \
  }

enum class FFTDims : uint {
  d1 = 1,
  d2 = 2,
  d3 = 3,
};

struct Parameters
{
  uint nGpus;
  FFTDims dims;
  dim3 sizes;

  void printSetup()
  {
    std::cout << "nGPUs: "     << nGpus
              << "\ndims: "    << static_cast<int>(dims)
              << "\nsizes.x: " << sizes.x
              << "\nsizes.y: " << sizes.y
              << "\nsizes.z: " << sizes.z
              <<  std::endl;
  }
};

Parameters parseArgs(int argc, const char* argv[])
{
  if (argc != 6)
  {
    std::cerr << "Invalid number of arguments" << std::endl;
    exit(1);
  }

  Parameters p;

  p.nGpus = static_cast<uint>(std::stoi(argv[1]));
  p.dims = static_cast<FFTDims>(std::stoi(argv[2]));
  p.sizes = {
    static_cast<uint>(std::stoi(argv[3])),
    static_cast<uint>(std::stoi(argv[4])),
    static_cast<uint>(std::stoi(argv[5]))
  };

  return p;
}

size_t getRSize(dim3 sizes)
{
  return sizes.x * sizes.y * sizes.z * sizeof(hipfftReal);
}

size_t getCSize(dim3 sizes)
{
  return (sizes.x * sizes.y * sizes.z / 2 + 1) * sizeof(hipfftComplex);
}

void init(const Parameters &p)
{
  int gpuCount;
  checkCudaError(hipGetDeviceCount(&gpuCount));

  if (gpuCount < p.nGpus)
  {
    std::cerr << "Not enough gpus" << std::endl;
    exit(1);
  }

  std::cout << "GPUs: " << p.nGpus << "/" << gpuCount << std::endl;
}

void initData(hipfftReal* dRs, dim3 sizes)
{
  hiprandGenerator_t generator;
  hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(generator, 0ull);

  hiprandGenerateUniform(generator, dRs, sizes.x * sizes.y * sizes.z);
}

hipfftHandle getPlan(const Parameters &params)
{
  std::vector<int> whichGPUs(params.nGpus);
  std::vector<size_t> workSize(params.nGpus);
  std::iota(whichGPUs.begin(), whichGPUs.end(), 0);

  hipfftHandle plan;
  checkCuFFTError(hipfftCreate(&plan));

  checkCuFFTError(hipfftXtSetGPUs(plan, params.nGpus, whichGPUs.data()));

  switch (params.dims)
  {
  case FFTDims::d1:
    checkCuFFTError(hipfftMakePlan1d(plan, params.sizes.x, HIPFFT_R2C, 1, workSize.data()));
    break;
  case FFTDims::d2:
    checkCuFFTError(hipfftMakePlan2d(plan, params.sizes.x, params.sizes.y, HIPFFT_R2C, workSize.data()));
    break;
  case FFTDims::d3:
    checkCuFFTError(hipfftMakePlan3d(plan, params.sizes.x, params.sizes.y, params.sizes.z, HIPFFT_R2C, workSize.data()));
    break;
  }

  std::cout << "Work sizes[" << workSize.size() << "]: ";
  for (auto i : workSize)
  {
    std::cout << i << ", ";
  }
  std::cout << std::endl;

  return plan;
}

void checkOutput(hipfftComplex* hCs)
{

}

int main(int argc, const char* argv[])
{
  Parameters params = parseArgs(argc, argv);
  params.printSetup();

  init(params);

  //  Allocate memory

  hipfftReal* hRs;
  hipfftComplex* hCs;

  hipfftReal* dRs;
  hipfftComplex* dCs;

  checkCudaError(hipHostAlloc(&hRs, getRSize(params.sizes), hipHostMallocMapped));
  checkCudaError(hipHostAlloc(&hCs, getCSize(params.sizes), hipHostMallocMapped));

  checkCudaError(hipMalloc(&dRs, getRSize(params.sizes)));
  checkCudaError(hipMalloc(&dCs, getCSize(params.sizes)));

  //  Create FFT plan

  hipfftHandle plan = getPlan(params);

  hipLibXtDesc* dRsXt;
  hipLibXtDesc* dCsXt;

  checkCuFFTError(hipfftXtMalloc(plan, &dRsXt, HIPFFT_XT_FORMAT_INPUT));
  checkCuFFTError(hipfftXtMalloc(plan, &dCsXt, HIPFFT_XT_FORMAT_OUTPUT));

  //  Initialize data

  initData(dRs, params.sizes);

  //  Copy initialized data from host to device

  checkCudaError(hipMemcpy(hRs, dRs, getRSize(params.sizes), hipMemcpyDeviceToHost));
  checkCuFFTError(hipfftXtMemcpy(plan, dRsXt, hRs, HIPFFT_COPY_HOST_TO_DEVICE));

  std::cout << "Data copied!" << std::endl;

  //  Execute computation

  auto start = std::chrono::high_resolution_clock::now();

  checkCuFFTError(hipfftXtExecDescriptorR2C(plan, dRsXt, dCsXt));
  checkCudaError(hipDeviceSynchronize());

  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::duration<float>>(end - start);

  std::cout << duration.count() << " s" << std::endl;

  //  Copy results from device to host

  // checkCudaError(cudaMemcpy(hCs, dCs, getCSize(params.sizes), cudaMemcpyDeviceToHost));
  checkCuFFTError(hipfftXtMemcpy(plan, hCs, dCsXt, HIPFFT_COPY_DEVICE_TO_HOST));

  //  Verify results

  checkOutput(dCs);

  //  Clean up

  checkCuFFTError(hipfftXtFree(dRsXt));
  checkCuFFTError(hipfftXtFree(dCsXt));
  checkCuFFTError(hipfftDestroy(plan));
  checkCudaError(hipFree(dRs));
  checkCudaError(hipFree(dCs));
  checkCudaError(hipHostFree(hRs));
  checkCudaError(hipHostFree(hCs));

  return 0;
}
