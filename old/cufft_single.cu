
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <string>

#include <hipfft/hipfft.h>
#include <hiprand/hiprand.h>
#include <hip/hip_runtime_api.h>

enum class FFTDims : uint {
  d1 = 1,
  d2 = 2,
  d3 = 3,
};

struct Parameters
{
  FFTDims dims;
  dim3 sizes;
};

Parameters parseArgs(int argc, const char* argv[])
{
  if (argc != 5)
  {
    std::cerr << "Invalid numer of arguments" << std::endl;
    exit(1);
  }

  Parameters p;

  p.dims = static_cast<FFTDims>(std::stoi(argv[1]));
  p.sizes = {
    static_cast<uint>(std::stoi(argv[2])),
    static_cast<uint>(std::stoi(argv[3])),
    static_cast<uint>(std::stoi(argv[4]))
  };

  return p;
}

size_t getRSize(dim3 sizes)
{
  return sizes.x * sizes.y * sizes.z * sizeof(hipfftReal);
}

size_t getCSize(dim3 sizes)
{
  return (sizes.x * sizes.y * sizes.z / 2 + 1) * sizeof(hipfftComplex);
}

hipfftHandle getPlan(Parameters &params)
{
  hipfftHandle plan;

  switch (params.dims)
  {
  case FFTDims::d1:
    hipfftPlan1d(&plan, params.sizes.x, HIPFFT_R2C, 1);
    break;
  case FFTDims::d2:
    hipfftPlan2d(&plan, params.sizes.x, params.sizes.y, HIPFFT_R2C);
    break;
  case FFTDims::d3:
    hipfftPlan3d(&plan, params.sizes.x, params.sizes.y, params.sizes.z, HIPFFT_R2C);
    break;
  }

  return plan;
}

void initData(hipfftReal* dRs, dim3 sizes)
{
  hiprandGenerator_t generator;
  hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(generator, 0ull);

  hiprandGenerateUniform(generator, dRs, sizes.x * sizes.y * sizes.z);
}

void checkOutput(hipfftComplex* hCs)
{

}

int main(int argc, const char* argv[])
{
  Parameters params = parseArgs(argc, argv);

  //  Allocate memory

  hipfftReal* hRs;
  hipfftComplex* hCs;

  hipfftReal* dRs;
  hipfftComplex* dCs;

  hipHostAlloc(&hRs, getRSize(params.sizes), hipHostMallocMapped);
  hipHostAlloc(&hCs, getCSize(params.sizes), hipHostMallocMapped);

  hipMalloc(&dRs,  getRSize(params.sizes));
  hipMalloc(&dCs, getCSize(params.sizes));

  //  Create FFT plan

  hipfftHandle plan = getPlan(params);

  //  Initialize data

  initData(dRs, params.sizes);

  //  Copy initialized data from host to device

  hipMemcpy(hRs, dRs, getRSize(params.sizes), hipMemcpyDeviceToHost);
  

  //  Execute computation

  auto start = std::chrono::high_resolution_clock::now();

  hipfftExecR2C(plan, dRs, dCs);
  hipDeviceSynchronize();

  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::duration<float>>(end - start);

  std::cout << duration.count() << " s" << std::endl;

  //  Copy results from device to host

  hipMemcpy(hCs, dCs, getCSize(params.sizes), hipMemcpyDeviceToHost);

  //  Verify results

  checkOutput(dCs);

  //  Clean up

  hipfftDestroy(plan);
  hipFree(dRs);
  hipFree(dCs);
  hipHostFree(hRs);
  hipHostFree(hCs);

  return 0;
}
