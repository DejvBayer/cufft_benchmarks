#include <chrono>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "gpu_utils.cuh"
#include "FFTSolver.cuh"

FFTSolver* FFTSolver::getSolver()
{
  Parameters& params = Parameters::getInstance();

  return params.getNGPUs() == 1
    ? reinterpret_cast<FFTSolver*>(new cufftSolver())
    : reinterpret_cast<FFTSolver*>(new cufftXtSolver());
}

FFTSolver::FFTSolver()
: mParams(Parameters::getInstance())
{}

void FFTSolver::compute(const void* input)
{}

cufftSolver::cufftSolver()
: FFTSolver(), mPlan(0), mInput(nullptr), mOutput(nullptr)
{
  switch (mParams.getNDims())
  {
  case Dimensions::d2:
    checkCufftError(hipfftPlan2d(&mPlan, mParams.getXDim(), mParams.getYDim(),
                    mParams.getCufftType()));
    break;
  case Dimensions::d3:
    checkCufftError(hipfftPlan3d(&mPlan, mParams.getXDim(), mParams.getYDim(),
                    mParams.getZDim(), mParams.getCufftType()));
    break;
  }

  checkCudaError(hipMalloc(&mInput, mParams.getInputSize()));
  checkCudaError(hipMalloc(&mOutput, mParams.getOutputSize()));
}

void cufftSolver::compute(const void* input)
{
  checkCudaError(hipMemcpy(mInput, input, mParams.getInputSize(), hipMemcpyHostToDevice));

  auto start = std::chrono::high_resolution_clock::now();

  switch (mParams.getFFTType())
  {
  case FFTType::C2C:
    checkCufftError(hipfftExecC2C(mPlan, reinterpret_cast<hipfftComplex*>(mInput),
                    reinterpret_cast<hipfftComplex*>(mOutput), HIPFFT_FORWARD));
    break;
  case FFTType::C2R:
    checkCufftError(hipfftExecC2R(mPlan, reinterpret_cast<hipfftComplex*>(mInput),
                    reinterpret_cast<hipfftReal*>(mOutput)));
    break;
  case FFTType::R2C:
    checkCufftError(hipfftExecR2C(mPlan, reinterpret_cast<hipfftReal*>(mInput),
                    reinterpret_cast<hipfftComplex*>(mOutput)));
    break;
  }
  checkCudaError(hipDeviceSynchronize());

  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::duration<float>>(end - start);

  std::cout << duration.count() << " s" << std::endl;
}

cufftSolver::~cufftSolver()
{
  hipfftDestroy(mPlan);
  hipFree(mInput);
  hipFree(mOutput);
}

cufftXtSolver::cufftXtSolver()
: FFTSolver(), mPlan(0), mInput(nullptr), mOutput(nullptr)
{
  std::vector<int> whichGPUs(mParams.getNGPUs());
  std::vector<size_t> workSize(mParams.getNGPUs());

  std::iota(whichGPUs.begin(), whichGPUs.end(), 0);

  checkCufftError(hipfftCreate(&mPlan));
  checkCufftError(hipfftXtSetGPUs(mPlan, mParams.getNGPUs(), whichGPUs.data()));

  switch (mParams.getNDims())
  {
  case Dimensions::d2:
    checkCufftError(hipfftMakePlan2d(mPlan, mParams.getXDim(), mParams.getYDim(),
                    mParams.getCufftType(), workSize.data()));
    break;
  case Dimensions::d3:
    checkCufftError(hipfftMakePlan3d(mPlan, mParams.getXDim(), mParams.getYDim(),
                    mParams.getZDim(), mParams.getCufftType(), workSize.data()));
    break;
  }

  checkCufftError(hipfftXtMalloc(mPlan, &mInput, HIPFFT_XT_FORMAT_INPLACE));
  checkCufftError(hipfftXtMalloc(mPlan, &mOutput, HIPFFT_XT_FORMAT_INPLACE));
}

void cufftXtSolver::compute(const void* input)
{
  checkCufftError(hipfftXtMemcpy(mPlan, mInput, const_cast<void*>(input), HIPFFT_COPY_HOST_TO_DEVICE));

  auto start = std::chrono::high_resolution_clock::now();

  switch (mParams.getFFTType())
  {
  case FFTType::C2C:
    checkCufftError(hipfftXtExecDescriptorC2C(mPlan, mInput, mOutput, HIPFFT_FORWARD));
    break;
  case FFTType::C2R:
    checkCufftError(hipfftXtExecDescriptorC2R(mPlan, mInput, mOutput));
    break;
  case FFTType::R2C:
    checkCufftError(hipfftXtExecDescriptorR2C(mPlan, mInput, mOutput));
    break;
  }

  checkCudaError(hipDeviceSynchronize());

  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::duration<float>>(end - start);

  std::cout << duration.count() << " s" << std::endl;
}

cufftXtSolver::~cufftXtSolver()
{
  hipfftDestroy(mPlan);
  hipfftXtFree(mInput);
  hipfftXtFree(mOutput);
}
