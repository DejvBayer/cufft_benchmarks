#include "Generator.h"
#include "gpu_utils.cuh"

Generator::Generator()
: mParams(Parameters::getInstance()), mGenerator(nullptr), mDevMem(nullptr), mHostMem(nullptr)
{
  checkCuRandError(hiprandCreateGenerator(&mGenerator, HIPRAND_RNG_PSEUDO_DEFAULT));
  checkCuRandError(hiprandSetPseudoRandomGeneratorSeed(mGenerator, sSeed));
}

void Generator::generate()
{
  if (mDevMem == nullptr)
  {
    size_t inputSize = mParams.getInputSize();
  
    checkCudaError(hipMalloc(&mDevMem, inputSize));
    checkCudaError(hipHostAlloc(&mHostMem, inputSize, hipHostMallocMapped));

    checkCuRandError(hiprandGenerateUniform(mGenerator, reinterpret_cast<float*>(mDevMem), mParams.getInputN()));

    checkCudaError(hipMemcpy(mHostMem, mDevMem, inputSize, hipMemcpyDeviceToHost));    
  }
}

Generator::~Generator()
{
  hiprandDestroyGenerator(mGenerator);
  hipFree(mDevMem);
  hipHostFree(mHostMem);
}
